#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "inc\hip/hip_runtime_api.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <string>
#include <algorithm>
#include <ctime>
#include <conio.h>

#include <tchar.h>
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""

#include <thrust/sort.h>

#include <stdio.h>
using namespace std;

///// DEFINE ///////
#define numberOfCities 100
const char * fileName = "kroa100.txt";
#define populationSize  4000          // wielko�� musi by� wielokrotno�ci� 4, inaczej krzy�owanie nie dzia�a do ko�ca poprawnie
#define iterations 10000

////// zmienne globalne ////////////////////////////////////////
int citiesArray[numberOfCities + 1][3];                       // miasta odczytane z pliku
int citiesDistance[(numberOfCities + 1)*(numberOfCities)];    // odleg�osci pomiedzy miastami 1D, +1 bo miasto o nr x ma miejsce w tablicy x
int populationArray[populationSize * numberOfCities];		  // tablica populacji 1D
////////////////////////////////////////////////////////////////

__constant__ int Distances[(numberOfCities + 1)*(numberOfCities)];


__global__ void CalculateDistance(const int* population, int* result)
{
	long tmpDistance = 0;
	int fromCity, toCity = 0;
	int tid = blockIdx.x;// *blockDim.x + threadIdx.x;
	int firstCity = tid * numberOfCities;
	for (int i = 0; i < numberOfCities - 1; i++)             ////////// odleg�osci od pierwszego do ostatniego
	{
		fromCity = population[ firstCity + i];
		toCity = population[ firstCity + i + 1];
		tmpDistance += Distances[ fromCity * numberOfCities + toCity];
	}
	fromCity = population[ (tid + 1) * numberOfCities - 1];        ///////// odleg�o�� z ostatniego do pocz�tkowego
	toCity = population[ firstCity];
	tmpDistance += Distances[ fromCity * numberOfCities + toCity];
	result[tid] = tmpDistance;
}

__global__ void reorderByKey(int* population, int* tmp_population, int* values)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int moveFrom = values[blockIdx.x] * numberOfCities;
	population[tid] = tmp_population[moveFrom + threadIdx.x];
}

__global__ void crossover(int *population)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = numberOfCities*populationSize / 2;
	if (blockIdx.x % 2 == 0)        // parzyste miasta
	{
		if (threadIdx.x < numberOfCities / 2)
			population[tid + offset] = population[tid];
		else
			population[tid + offset + numberOfCities] = population[tid];
	}
	else
	{
		if (threadIdx.x < numberOfCities / 2)
			population[tid + offset] = population[tid];
		else
			population[tid + offset - numberOfCities] = population[tid];

	}
}

__global__ void normalizacja(int *population)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (populationSize)
	{
		int tempArray[numberOfCities] = { 0 }; 
		int id = tid*numberOfCities;
		int city = 0;
		int counter = 0;
		for (int i = 0; i < numberOfCities; i++)
		{
			city = population[id+i];
			if (tempArray[city] == 0)
				tempArray[city] = 1;
			else
			{
				population[id+i] = -1;
				counter++;
			}
		}
		for (int i = 0; i < counter; i++)
		{
			int PositionA = 0, PositionB = 0;
			for (int j = PositionA; j < numberOfCities ; j++)
			{
				if (tempArray[j] == 0)
				{
					PositionA = j;
					tempArray[j] = 1;
					break;
				}
			}
			for (int j = PositionB; j < numberOfCities; j++)
			{
				if (population[id + j] == -1)
				{
					population[id + j] = PositionA;
					PositionB = j;
					break;
				}
			}
		}

	}
	

	for (int i = 0; i < numberOfCities; i++)
	{
		//int city = populationArray[]     
		/*if (tempArray[i] == 0)
			tempArray[i] = 1;
		else*/
			
	}
}

///// prototypy funkcji /////////////////
int* cudaCalculateResult(int *, int );
int* gpuSort(int*, int **);
void findDuplicates(int*);

void fillCitiesArray()
{
	ifstream fin;
	fin.open(fileName);
	if (fin.is_open())
	{
		string currentLine;
		string tmp;


		citiesArray[0][0] = 0;
		citiesArray[0][1] = 0;
		citiesArray[0][2] = 0;

		int count = 1;
		while (count < numberOfCities + 1)

		{
			fin >> citiesArray[count][0];
			fin >> citiesArray[count][1];
			fin >> citiesArray[count][2];
			count++;
		}
	}
}

void fillCitiesDistance()
{
	int distance, disX, disY;
	citiesDistance[0] = 0;        // p�tla nie obejmuje, niepotrzebne ustawione na 0
	for (int i = 1; i < numberOfCities + 1; i++)
	{
		citiesDistance[i] = 0;   //niepotrzebne kom�rki ustawione na 0
		citiesDistance[i*numberOfCities] = 0;
		for (int j = 1; j < numberOfCities; j++)
		{
			disX = _Pow_int(citiesArray[i][1] - citiesArray[j][1], 2);
			disY = _Pow_int(citiesArray[i][2] - citiesArray[j][2], 2);
			distance = (int)sqrt(disX + disY);
			citiesDistance[i*(numberOfCities)+j] = distance;
		}
	}
}

int myRandom(int i) { return std::rand() % i; }

void addGenom(int position)
{
	int idx = position * numberOfCities;
	for (int i = 0; i < numberOfCities; i++)
	{
		populationArray[idx + i] = i + 1;  // wpisanie numer�w miast do genomu
	}
		random_shuffle(&populationArray[idx], &populationArray[idx + numberOfCities], myRandom);
		
}

void mutatePopulation()
{
	for (int i = 1; i < populationSize; i++)
	{
		int rand1 = std::rand()%numberOfCities;
		int rand2;
		do
		{
			rand2 = rand()%numberOfCities;
		} while (rand1==rand2);
		int id = i*numberOfCities;
		int temp = populationArray[id + rand1];
		populationArray[id + rand1] = populationArray[id + rand2];
		populationArray[id + rand2] = temp;
	}
}

void cudaInitialization()
{
	hipError_t cudaSucces = hipMemcpyToSymbol(HIP_SYMBOL(Distances), citiesDistance, sizeof(citiesDistance));
	if (cudaSucces != hipSuccess)
	{
		printf("Blad kopiowania do pamieci gpu");
	}
}    // kopiuje odleg�o��i pomi�dzy miastami do pami�ci sta�ej GPU

int main()
{
	std::srand(unsigned(std::time(0)));
	fillCitiesArray();
	fillCitiesDistance();
	for (int i = 0; i < populationSize; i++)
	{
		addGenom(i);
	}
	cudaInitialization();
	/// inicjalizacja zako�czona

	int result[populationSize];
	
	cudaCalculateResult(result, iterations);
	
	cout << result[0] << endl;
	return 0;
}

//void normalizacja()
//{
//	for (int i = populationSize/2; i < populationSize; i++)         //sprawdzenie konieczne jedynie dla drugiej cz�ci populacji
//	{
//		int tmpCity = 0;
//		int temp[numberOfCities];
//		
//		for (int j = 0; j < numberOfCities; j++)
//		{
//			temp[j] = -2;                            // -2 oznacza brak w miasta w tablicy
//		}
//
//		int id = i * numberOfCities;                // pocz�tek genomu w tablicy populacji
//		for (int k = 0; k < numberOfCities; k++)
//		{
//			tmpCity = populationArray[id + k];
//			temp[tmpCity-1] == -2 ? temp[tmpCity-1] = -1 : temp[tmpCity-1] = k;
//		} // dzia�a ok do tego miejsca
//		for (int i = 0; i < numberOfCities; i++)
//		{
//			int position = 0;
//			for (int j = 0; j < numberOfCities; j++)
//			{
//				if (temp[j]==-2)
//				{
//					tmpCity = j + 1;
//					break;
//				}
//			}
//			for (int k = 0; k < numberOfCities; k++)
//			{
//				if (temp[k]>-1)
//				{
//					populationArray[id + temp[k]] = tmpCity;
//					temp[tmpCity-1] = -1;
//					temp[k] = -1;
//					break;
//				}
//			}
//		}
//	}
//}

int* cudaCalculateResult(int * result, int iter)
{
	static int firstUse;
	static int keys[populationSize];
	static int * dev_populationArray;
	static int * dev_result;
	static int * p_tmp;

	if (!firstUse)
	{
		checkCudaErrors(hipMalloc((void **)&dev_populationArray, sizeof(populationArray)));
		checkCudaErrors(hipMalloc((void**)&dev_result, populationSize*sizeof(int)));
		firstUse = 1;
	}
	////////////////////////////////// g�owna p�tla programu
	for (int i = 0; i < iter; i++)
	{
		checkCudaErrors(hipMemcpy(dev_populationArray, populationArray, sizeof(populationArray), hipMemcpyHostToDevice));

		CalculateDistance << <populationSize, 1 >> >(dev_populationArray, dev_result);
		hipMemcpy(keys, dev_result, populationSize*sizeof(int), hipMemcpyDeviceToHost);

		p_tmp = gpuSort(keys, &dev_populationArray);

		crossover << <populationSize / 2, numberOfCities >> >(dev_populationArray);
		hipDeviceSynchronize();

		normalizacja << <(populationSize + 127) / 128, 128 >> >(dev_populationArray);
		hipMemcpy(populationArray, dev_populationArray, sizeof(populationArray), hipMemcpyDeviceToHost);

		if (i%10==0)
		findDuplicates(keys);

		mutatePopulation();

		cout << keys[0] << endl;
	}
	///////////////////////////////////// kopiowanie wynikow "na zewnatrz" i finalizacja
	for (int i = 0; i < populationSize; i++)
	{
		result[i] = keys[i];
	}
	
	hipFree(dev_populationArray);
	hipFree(dev_result);
	hipFree(p_tmp);
	return result;
}

int* gpuSort(int* keys, int **popAray)   // keys - oceny populacji
{
	static int* dev_values;
	static int noFirst;
	static int* tmp_population;
	static int values[populationSize];

	for (int i = 0; i < populationSize; i++)
	{
		values[i] = i;
	}
	
	thrust::sort_by_key(keys, keys + populationSize, values);

	if (!noFirst)
	{
		checkCudaErrors(hipMalloc((void**)&tmp_population, sizeof(populationArray)));
		checkCudaErrors(hipMalloc((void**)&dev_values, populationSize*sizeof(int)));
		noFirst = 1;
	}
	
	checkCudaErrors(hipMemcpy(dev_values, values, populationSize*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(tmp_population, *popAray, sizeof(populationArray), hipMemcpyDeviceToDevice));
	hipDeviceSynchronize();

	reorderByKey << <populationSize, numberOfCities >> >(*popAray, tmp_population, dev_values); 
	hipDeviceSynchronize();
	return tmp_population;
}

void findDuplicates(int* keys)
{
	for (int i = 0; i < populationSize-1; i++)
	{
		if (keys[i] == keys[i + 1])
		{
			addGenom(i + 1);
		}
	}
}

